#include "hip/hip_runtime.h"
/*
 * DCTHEVCdct.cu
 *
 *  Created on: 5 Mar 2016
 *      Author: vegan
 */

#include <stdlib.h>
#include <algorithm>
#include <limits>
#include <assert.h>

#include "DCTHEVCdct.hpp"
#include "DCTHEVC.hpp"

#include <device_functions.hpp> // nv min, max

#define MATRIX_MULT true

#if MATRIX_MULT
/** NxN forward transform (2D) using brute force matrix multiplication (3 nested loops)
 *  \param block pointer to input data (residual)
 *  \param coeff pointer to output data (transform coefficients)
 *  \param uiStride stride of input data
 *  \param uiTrSize transform size (uiTrSize x uiTrSize)
 *  \param uiMode is Intra Prediction mode used in Mode-Dependent DCT/DST only
 */
Void xTr(Int bitDepth, Pel *block, TCoeff *coeff, UInt uiStride, UInt uiTrSize, Bool useDST, const Int maxLog2TrDynamicRange)
{
	/*
	Mostly HEVC code
	*/
	UInt i,j,k;
	TCoeff iSum;
	TCoeff tmp[MAX_TU_SIZE * MAX_TU_SIZE];
	const TMatrixCoeff *iT;
	const SChar		g_aucConvertToBit[ MAX_CU_SIZE+1 ] =
	{\
			-1, -1, -1, -1,  0, -1, -1, -1, \
			1, -1, -1, -1, -1, -1, -1, -1, \
			2, -1, -1, -1, -1, -1, -1, -1, \
			-1, -1, -1, -1, -1, -1, -1, -1, \
			3, -1, -1, -1, -1, -1, -1, -1, \
			-1, -1, -1, -1, -1, -1, -1, -1, \
			-1, -1, -1, -1, -1, -1, -1, -1, \
			-1, -1, -1, -1, -1, -1, -1, -1, \
			4};   // from width to log2(width)-2
	UInt uiLog2TrSize = g_aucConvertToBit[ uiTrSize ] + 2;

	if (uiTrSize==4)
	{
		//    iT  = (useDST ? g_as_DST_MAT_4[TRANSFORM_FORWARD][0] : g_aiT4[TRANSFORM_FORWARD][0]);
		iT  = g_aiT4[TRANSFORM_FORWARD][0];
	}
	else if (uiTrSize==8)
	{
		iT = g_aiT8[TRANSFORM_FORWARD][0];
	}
	else if (uiTrSize==16)
	{
		iT = g_aiT16[TRANSFORM_FORWARD][0];
	}
	else if (uiTrSize==32)
	{
		iT = g_aiT32[TRANSFORM_FORWARD][0];
	}
	else
	{
		assert(0);
	}

	const Int TRANSFORM_MATRIX_SHIFT = g_transformMatrixShift[TRANSFORM_FORWARD];

	const Int shift_1st = (uiLog2TrSize +  bitDepth + TRANSFORM_MATRIX_SHIFT) - maxLog2TrDynamicRange;
	const Int shift_2nd = uiLog2TrSize + TRANSFORM_MATRIX_SHIFT;
	const Int add_1st = (shift_1st>0) ? (1<<(shift_1st-1)) : 0;
	const Int add_2nd = 1<<(shift_2nd-1);

	/* Horizontal transform */

	for (i=0; i<uiTrSize; i++)
	{
		for (j=0; j<uiTrSize; j++)
		{
			iSum = 0;
			for (k=0; k<uiTrSize; k++)
			{
				iSum += iT[i*uiTrSize+k]*block[j*uiStride+k];
			}
			tmp[i*uiTrSize+j] = (iSum + add_1st)>>shift_1st;
		}
	}

	/* Vertical transform */
	for (i=0; i<uiTrSize; i++)
	{
		for (j=0; j<uiTrSize; j++)
		{
			iSum = 0;
			for (k=0; k<uiTrSize; k++)
			{
				iSum += iT[i*uiTrSize+k]*tmp[j*uiTrSize+k];
			}
			coeff[i*uiTrSize+j] = (iSum + add_2nd)>>shift_2nd;
		}
	}
}

/** NxN inverse transform (2D) using brute force matrix multiplication (3 nested loops)
 *  \param coeff pointer to input data (transform coefficients)
 *  \param block pointer to output data (residual)
 *  \param uiStride stride of output data
 *  \param uiTrSize transform size (uiTrSize x uiTrSize)
 *  \param uiMode is Intra Prediction mode used in Mode-Dependent DCT/DST only
 */
Void xITr(Int bitDepth, TCoeff *coeff, Pel *block, UInt uiStride, UInt uiTrSize, Bool useDST, const Int maxLog2TrDynamicRange)
{
	/*
	Mostly HEVC code
	*/
	UInt i,j,k;
	TCoeff iSum;
	TCoeff tmp[MAX_TU_SIZE * MAX_TU_SIZE];
	const TMatrixCoeff *iT;

	if (uiTrSize==4)
	{
		//    iT  = (useDST ? g_as_DST_MAT_4[TRANSFORM_INVERSE][0] : g_aiT4[TRANSFORM_INVERSE][0]);
		iT  = g_aiT4[TRANSFORM_INVERSE][0];
	}
	else if (uiTrSize==8)
	{
		iT = g_aiT8[TRANSFORM_INVERSE][0];
	}
	else if (uiTrSize==16)
	{
		iT = g_aiT16[TRANSFORM_INVERSE][0];
	}
	else if (uiTrSize==32)
	{
		iT = g_aiT32[TRANSFORM_INVERSE][0];
	}
	else
	{
		assert(0);
	}

	const Int TRANSFORM_MATRIX_SHIFT = g_transformMatrixShift[TRANSFORM_INVERSE];

	const Int shift_1st = TRANSFORM_MATRIX_SHIFT + 1; //1 has been added to shift_1st at the expense of shift_2nd
	const Int shift_2nd = (TRANSFORM_MATRIX_SHIFT + maxLog2TrDynamicRange - 1) - bitDepth;
	const TCoeff clipMinimum = -(1 << maxLog2TrDynamicRange);
	const TCoeff clipMaximum =  (1 << maxLog2TrDynamicRange) - 1;
	assert(shift_2nd>=0);
	const Int add_1st = 1<<(shift_1st-1);
	const Int add_2nd = (shift_2nd>0) ? (1<<(shift_2nd-1)) : 0;

	/* Horizontal transform */
	for (i=0; i<uiTrSize; i++)
	{
		for (j=0; j<uiTrSize; j++)
		{
			iSum = 0;
			for (k=0; k<uiTrSize; k++)
			{
				iSum += iT[k*uiTrSize+i]*coeff[k*uiTrSize+j];
			}

			// Clipping here is not in the standard, but is used to protect the "Pel" data type into which the inverse-transformed samples will be copied
			tmp[i*uiTrSize+j] = Clip3<TCoeff>(clipMinimum, clipMaximum, (iSum + add_1st)>>shift_1st);
		}
	}

	/* Vertical transform */
	for (i=0; i<uiTrSize; i++)
	{
		for (j=0; j<uiTrSize; j++)
		{
			iSum = 0;
			for (k=0; k<uiTrSize; k++)
			{
				iSum += iT[k*uiTrSize+j]*tmp[i*uiTrSize+k];
			}

			block[i*uiStride+j] = Clip3<TCoeff>(std::numeric_limits<Pel>::min(), std::numeric_limits<Pel>::max(), (iSum + add_2nd)>>shift_2nd);
		}
	}
}

#endif //MATRIX_MULT

/*
Threaded code
*/
__device__ Void partialButterfly4ThreadTwo(\
		TCoeff*	src, \
		TCoeff*	dst, \
		Int 	shift, \
		Int 	line, \
		TCoeff 	add, \
		Int 	k, \
		TMatrixCoeff *g_aiT_GPU)
{
	TCoeff a = *src;
	TCoeff b = *(src+1);
	*dst = (*(g_aiT_GPU+4*k)*a + *(g_aiT_GPU+4*k+1)*b + add)>>shift;
}

__device__ Void partialButterfly4ThreadOneP(\
		TCoeff*	src1, \
		TCoeff*	src2, \
		TCoeff*	dst)
{
	*dst = *src1 + *src2;
}

__device__ Void partialButterfly4ThreadOneN(\
		TCoeff*	src1, \
		TCoeff*	src2, \
		TCoeff*	dst)
{
	*dst = *src1 - *src2;
}

__device__ Void partialButterfly4ThreadZero(\
		TCoeff*			src, \
		TCoeff*			dst, \
		Int 			shift, \
		Int 			line, \
		const TCoeff 	add, \
		TMatrixCoeff *g_aiT_GPU)
{
	TCoeff E[2],O[2];

	partialButterfly4ThreadOneP(src, 	src+3, E);
	partialButterfly4ThreadOneN(src, 	src+3, O);
	partialButterfly4ThreadOneP(src+1, 	src+2, E+1);
	partialButterfly4ThreadOneN(src+1, 	src+2, O+1);

	partialButterfly4ThreadTwo(E, dst, 		shift, line, add, 0, g_aiT_GPU);
	partialButterfly4ThreadTwo(O, dst + 1*line, 	shift, line, add, 1, g_aiT_GPU);
	partialButterfly4ThreadTwo(E, dst + 2*line,	shift, line, add, 2, g_aiT_GPU);
	partialButterfly4ThreadTwo(O, dst + 3*line, 	shift, line, add, 3, g_aiT_GPU);
}

/** 4x4 forward transform implemented using partial butterfly structure (1D)
 *  \param src   input data (residual)
 *  \param dst   output data (transform coefficients)
 *  \param shift specifies right shift after 1D transform
 *  \param line
 */
__global__ Void partialButterfly4(\
		TCoeff*	src, \
		TCoeff*	dst, \
		Int 	shift, \
		Int 	line, \
		TMatrixCoeff *g_aiT_GPU)
{
	__shared__ TCoeff add;
	Int j = blockIdx.x * blockDim.x + threadIdx.x;
	__syncthreads();

	if(j == 0) add = (shift > 0) ? (1<<(shift-1)) : 0;
	__syncthreads();

	if(j < 4)
	{
		partialButterfly4ThreadZero(src+4*j, dst+j, shift, line, add, g_aiT_GPU);
	}
}

__device__ Void Clip3C (\
		TCoeff*			dst, \
		const TCoeff 	minVal, \
		const TCoeff 	maxVal, \
		const TCoeff 	a)
{
	*dst = min(max(minVal, a), maxVal);
}

__device__ Void partialButterflyInverse4ThreadTwo(\
		TCoeff 			E, \
		TCoeff			O, \
		TCoeff*			dst, \
		Int 			shift, \
		const TCoeff 	outputMinimum, \
		const TCoeff 	outputMaximum, \
		TCoeff 			add)
{
	Clip3C(dst, outputMinimum, outputMaximum, (E + O + add)>>shift );
	//*dst = Clip3( outputMinimum, outputMaximum, (E + O + add)>>shift );
}

__device__ Void partialButterflyInverse4ThreadEO(\
		TCoeff*	src1, \
		TCoeff*	src2, \
		TCoeff*	dst, \
		Int 	indA, \
		Int 	indB, \
		Int 	indx, \
		TMatrixCoeff *g_aiT_GPU)
{
	*dst = *(g_aiT_GPU+4*indA+indx)*(*src1) + *(g_aiT_GPU+4*indB+indx)*(*src2);
}


__device__ Void partialButterflyInverse4ThreadZero(\
		TCoeff*			src, \
		TCoeff*			dst, \
		Int 			shift, \
		Int 			line, \
		const TCoeff 	outputMinimum, \
		const TCoeff 	outputMaximum, \
		TCoeff 			add, \
		TMatrixCoeff *g_aiT_GPU)
{
	TCoeff E[2],O[2];

	partialButterflyInverse4ThreadEO(src+line, 	src+3*line, O, 		1, 3, 0, g_aiT_GPU);
	partialButterflyInverse4ThreadEO(src+line, 	src+3*line, O+1, 	1, 3, 1, g_aiT_GPU);
	partialButterflyInverse4ThreadEO(src, 		src+2*line, E, 		0, 2, 0, g_aiT_GPU);
	partialButterflyInverse4ThreadEO(src, 		src+2*line, E+1, 	0, 2, 1, g_aiT_GPU);

	partialButterflyInverse4ThreadTwo(E[0],  O[0], dst, 	shift, outputMinimum, outputMaximum, add);
	partialButterflyInverse4ThreadTwo(E[1],  O[1], dst+1, 	shift, outputMinimum, outputMaximum, add);
	partialButterflyInverse4ThreadTwo(E[1], -O[1], dst+2, 	shift, outputMinimum, outputMaximum, add);
	partialButterflyInverse4ThreadTwo(E[0], -O[0], dst+3, 	shift, outputMinimum, outputMaximum, add);
}

/** 4x4 inverse transform implemented using partial butterfly structure (1D)
 *  \param src   input data (transform coefficients)
 *  \param dst   output data (residual)
 *  \param shift specifies right shift after 1D transform
 *  \param line
 *  \param outputMinimum  minimum for clipping
 *  \param outputMaximum  maximum for clipping
 */
__global__ Void partialButterflyInverse4(\
		TCoeff*			src, \
		TCoeff*			dst, \
		Int 			shift, \
		Int 			line, \
		const TCoeff 	outputMinimum, \
		const TCoeff 	outputMaximum, \
		TMatrixCoeff *g_aiT_GPU)
{
	__shared__ TCoeff add;
	Int j = blockIdx.x * blockDim.x + threadIdx.x;
	__syncthreads();

	if(j == 0) add = (shift > 0) ? (1<<(shift-1)) : 0;
	__syncthreads();

	if(j < 4)
	{
		partialButterflyInverse4ThreadZero(src+j, dst+4*j, shift, line, outputMinimum, outputMaximum, add, g_aiT_GPU);
	}
}




__device__ Void partialButterfly8ThreadOneB(\
		TCoeff*	src, \
		TCoeff*	dst, \
		Int 	shift, \
		Int 	line, \
		Int 	add, \
		Int 	indx, \
		TMatrixCoeff *g_aiT_GPU)
{
	*dst = (*(g_aiT_GPU+8*indx)*(*(src)) + *(g_aiT_GPU+8*indx+1)*(*(src+1)) + *(g_aiT_GPU+8*indx+2)*(*(src+2)) + *(g_aiT_GPU+8*indx+3)*(*(src+3)) + add)>>shift;
}

__device__ Void partialButterfly8ThreadOneA(\
		TCoeff*	src, \
		TCoeff*	dst, \
		Int 	shift, \
		Int 	line, \
		Int 	add, \
		Int 	indx, \
		TMatrixCoeff *g_aiT_GPU)
{
	*dst = (*(g_aiT_GPU+8*indx)*(*(src)) + *(g_aiT_GPU+8*indx+1)*(*(src+1)) + add)>>shift;
}

__device__ Void partialButterfly8ThreadZero(\
		TCoeff*			src, \
		TCoeff*			dst, \
		Int 			shift, \
		Int 			line, \
		const TCoeff 	add, \
		TMatrixCoeff *g_aiT_GPU)
{
	Int k;
	TCoeff E[4],O[4];
	TCoeff EE[2],EO[2];

	/* E and O*/
	for (k=0;k<4;k++)
	{
		partialButterfly4ThreadOneP(src+k, src+7-k, E + k);
		partialButterfly4ThreadOneN(src+k, src+7-k, O + k);
	}

	/* EE and EO */

	partialButterfly4ThreadOneP(E, 		E + 3, EE);
	partialButterfly4ThreadOneN(E, 		E + 3, EO);
	partialButterfly4ThreadOneP(E+1, 	E+2, EE+1);
	partialButterfly4ThreadOneN(E+1, 	E+2, EO+1);

	partialButterfly8ThreadOneA(EE, dst, 		shift, line, add, 0, g_aiT_GPU);
	partialButterfly8ThreadOneA(EE, dst+4*line, 	shift, line, add, 4, g_aiT_GPU);
	partialButterfly8ThreadOneA(EO, dst+2*line, 	shift, line, add, 2, g_aiT_GPU);
	partialButterfly8ThreadOneA(EO, dst+6*line, 	shift, line, add, 6, g_aiT_GPU);

	partialButterfly8ThreadOneB(O, dst+line, 	shift, line, add, 1, g_aiT_GPU);
	partialButterfly8ThreadOneB(O, dst+3*line, 	shift, line, add, 3, g_aiT_GPU);
	partialButterfly8ThreadOneB(O, dst+5*line, 	shift, line, add, 5, g_aiT_GPU);
	partialButterfly8ThreadOneB(O, dst+7*line, 	shift, line, add, 7, g_aiT_GPU);
}

/** 8x8 forward transform implemented using partial butterfly structure (1D)
 *  \param src   input data (residual)
 *  \param dst   output data (transform coefficients)
 *  \param shift specifies right shift after 1D transform
 *  \param line
 */
__global__ Void partialButterfly8(\
		TCoeff*	src, \
		TCoeff*	dst, \
		Int 	shift, \
		Int 	line, \
		TMatrixCoeff *g_aiT_GPU)
{
	__shared__ TCoeff add;
	Int j = blockIdx.x * blockDim.x + threadIdx.x;
	__syncthreads();

	if(j == 0) add = (shift > 0) ? (1<<(shift-1)) : 0;
	__syncthreads();

	if(j < 8)
	{
		partialButterfly8ThreadZero(src+8*j, dst+j, shift, line, add, g_aiT_GPU);
	}
}



__device__ Void partialButterflyInverse8threadZeroA(\
		TCoeff*	src, \
		Int 	line, \
		Int 	k,
		TCoeff*	O, \
		TMatrixCoeff *g_aiT_GPU)
{
	*O = 		*(g_aiT_GPU+8 +k)*(*(src+  line)) + *(g_aiT_GPU+24+k)*(*(src+3*line)) +
			*(g_aiT_GPU+40+k)*(*(src+5*line)) + *(g_aiT_GPU+56+k)*(*(src+7*line));
}

__device__ Void partialButterflyInverse8threadZeroB(\
		TCoeff*	src, \
		Int 	line, \
		Int 	ai, \
		Int 	aj, \
		Int 	bi, \
		TCoeff*	dst, \
		TMatrixCoeff *g_aiT_GPU)
{
	*(dst+aj) = *(g_aiT_GPU+8*ai+aj)*(*(src+line*ai)) + *(g_aiT_GPU+8*bi+aj)*(*(src+line*bi));
}

/*
__device__ Void partialButterflyInverse8threadOne(\
		TCoeff*			dst, \
		TCoeff*			E, \
		TCoeff*			O, \
		Int 			shift, \
		Int 			k, \
		Int 			s, \
		const TCoeff 	outputMinimum, \
		const TCoeff 	outputMaximum, \
		TCoeff 			add)
{
	*dst = Clip3( outputMinimum, outputMaximum, (*(E+k) + s*(*(O+k)) + add)>>shift );
}
*/

__device__ Void partialButterflyInverse8threadZero(\
		TCoeff*			src, \
		TCoeff*			dst, \
		Int 			shift, \
		Int 			line, \
		const TCoeff 	outputMinimum, \
		const TCoeff 	outputMaximum, \
		TCoeff 			add, \
		TMatrixCoeff *g_aiT_GPU)
{
	Int k;

	TCoeff E[4],O[4];
	TCoeff EE[2],EO[2];

	/* Utilizing symmetry properties to the maximum to minimize the number of multiplications */
	for (k=0;k<4;k++)
	{
		partialButterflyInverse8threadZeroA(src, line, k, O + k, g_aiT_GPU);
	}

	partialButterflyInverse8threadZeroB(src, line, 2, 0, 6, EO, 	g_aiT_GPU);
	partialButterflyInverse8threadZeroB(src, line, 2, 1, 6, EO, 	g_aiT_GPU);
	partialButterflyInverse8threadZeroB(src, line, 0, 0, 4, EE, 	g_aiT_GPU);
	partialButterflyInverse8threadZeroB(src, line, 0, 1, 4, EE, 	g_aiT_GPU);

	/* Combining even and odd terms at each hierarchy levels to calculate the final spatial domain vector */
	partialButterfly4ThreadOneP(EE, 	EO, 	E);
	partialButterfly4ThreadOneN(EE, 	EO, 	E+3);
	partialButterfly4ThreadOneP(EE+1, 	EO+1, 	E+1);
	partialButterfly4ThreadOneN(EE+1, 	EO+1, 	E+2);

	for (k=0;k<4;k++)
	{
		partialButterflyInverse4ThreadTwo(E[k], 	 O[k], 		dst+k, 		shift, outputMinimum, outputMaximum, add);
		partialButterflyInverse4ThreadTwo(E[3-k], 	-O[3-k], 	dst+k+4, 	shift, outputMinimum, outputMaximum, add);
	}
}

/** 8x8 inverse transform implemented using partial butterfly structure (1D)
 *  \param src   input data (transform coefficients)
 *  \param dst   output data (residual)
 *  \param shift specifies right shift after 1D transform
 *  \param line
 *  \param outputMinimum  minimum for clipping
 *  \param outputMaximum  maximum for clipping
 */
__global__ Void partialButterflyInverse8(
		TCoeff*			src, \
		TCoeff*			dst, \
		Int 			shift, \
		Int 			line, \
		const TCoeff 	outputMinimum, \
		const TCoeff 	outputMaximum, \
		TMatrixCoeff *g_aiT_GPU)
{
	__shared__ TCoeff add;
	Int j = blockIdx.x * blockDim.x + threadIdx.x;
	__syncthreads();

	if(j == 0) add = (shift > 0) ? (1<<(shift-1)) : 0;
	__syncthreads();

	if(j < 8)
	{
		partialButterflyInverse8threadZero(src+j, dst+8*j, shift, line, outputMinimum, outputMaximum, add, g_aiT_GPU);
	}
}



__device__ Void partialButterfly16threadZeroA(\
		TCoeff*	src, \
		Int 	ai, \
		Int 	aj, \
		Int 	si, \
		Int 	bi, \
		Int 	bj, \
		Int 	sj, \
		TCoeff 	add, \
		Int 	shift, \
		TCoeff*	dst, \
		TMatrixCoeff *g_aiT_GPU)
{
	*dst = (*(g_aiT_GPU+16*ai+aj)*(*(src+si)) + *(g_aiT_GPU+16*bi+bj)*(*(src+sj)) + add)>>shift;
}

__device__ Void partialButterfly16threadZeroB(\
		TCoeff*	src, \
		Int 	k, \
		TCoeff 	add, \
		Int 	shift, \
		TCoeff*	dst, \
		TMatrixCoeff *g_aiT_GPU)
{
	*dst = (*(g_aiT_GPU+16*k)*(*(src)) + *(g_aiT_GPU+16*k+1)*(*(src+1)) +
			*(g_aiT_GPU+16*k+2)*(*(src+2)) + *(g_aiT_GPU+16*k+3)*(*(src+3)) + add)>>shift;
}

__device__ Void partialButterfly16threadZeroC(\
		TCoeff*	src, \
		Int 	k, \
		TCoeff 	add, \
		Int 	shift, \
		TCoeff*	dst, \
		TMatrixCoeff *g_aiT_GPU)
{
	*dst = (*(g_aiT_GPU+16*k)*(*(src)) + *(g_aiT_GPU+16*k+1)*(*(src+1)) +
			*(g_aiT_GPU+16*k+2)*(*(src+2)) + *(g_aiT_GPU+16*k+3)*(*(src+3)) +
			*(g_aiT_GPU+16*k+4)*(*(src+4)) + *(g_aiT_GPU+16*k+5)*(*(src+5)) +
			*(g_aiT_GPU+16*k+6)*(*(src+6)) + *(g_aiT_GPU+16*k+7)*(*(src+7)) + add)>>shift;
}

__device__ Void partialButterfly16threadZero(\
		TCoeff*	src, \
		TCoeff*	dst, \
		Int 	shift, \
		Int 	line, \
		TCoeff 	add, \
		TMatrixCoeff *g_aiT_GPU)
{
	Int k;
	TCoeff E[8],O[8];
	TCoeff EE[4],EO[4];
	TCoeff EEE[2],EEO[2];
	/* E and O*/
	for (k=0;k<8;k++)
	{
		partialButterfly4ThreadOneP(src+k, 	src+15-k, 	E+k);
		partialButterfly4ThreadOneN(src+k, 	src+15-k, 	O+k);
	}

	/* EE and EO */
	for (k=0;k<4;k++)
	{
		partialButterfly4ThreadOneP(E+k, 	E+7-k, 	EE+k);
		partialButterfly4ThreadOneN(E+k, 	E+7-k, 	EO+k);
	}

	/* EEE and EEO */

	partialButterfly4ThreadOneP(EE, 	EE+3, 	EEE);
	partialButterfly4ThreadOneN(EE, 	EE+3, 	EEO);
	partialButterfly4ThreadOneP(EE+1, 	EE+2, 	EEE+1);
	partialButterfly4ThreadOneN(EE+1, 	EE+2, 	EEO+1);

	partialButterfly16threadZeroA(EEE, 0, 	0, 0, 0, 	1, 1, add, shift, dst, 		g_aiT_GPU);
	partialButterfly16threadZeroA(EEE, 8, 	0, 0, 8, 	1, 1, add, shift, dst+ 8*line, 	g_aiT_GPU);
	partialButterfly16threadZeroA(EEO, 4, 	0, 0, 4, 	1, 1, add, shift, dst+4*line, 	g_aiT_GPU);
	partialButterfly16threadZeroA(EEO, 12, 	0, 0, 12, 	1, 1, add, shift, dst+12*line, 	g_aiT_GPU);

	for (k=2;k<16;k+=4)
	{
		partialButterfly16threadZeroB(EO, k, add, shift, dst+k*line, g_aiT_GPU);
	}

	for (k=1;k<16;k+=2)
	{
		partialButterfly16threadZeroC(O, k, add, shift, dst+k*line, g_aiT_GPU);
	}
}

/** 16x16 forward transform implemented using partial butterfly structure (1D)
 *  \param src   input data (residual)
 *  \param dst   output data (transform coefficients)
 *  \param shift specifies right shift after 1D transform
 *  \param line
 */
__global__ Void partialButterfly16(\
		TCoeff*	src, \
		TCoeff*	dst, \
		Int 	shift, \
		Int 	line, \
		TMatrixCoeff *g_aiT_GPU)
{
	__shared__ TCoeff add;
	Int j = blockIdx.x * blockDim.x + threadIdx.x;
	__syncthreads();

	if(j == 0) add = (shift > 0) ? (1<<(shift-1)) : 0;
	__syncthreads();

	if(j < 16)
	{
		partialButterfly16threadZero(src+16*j, dst+j, shift, line, add, g_aiT_GPU);
	}
}


__device__ Void partialButterflyInverse16threadZeroA(\
		TCoeff* 	src, \
		Int 		line, \
		Int		k, \
		TCoeff* 	dst, \
		TMatrixCoeff *g_aiT_GPU)
{
	*(dst+k) = 	*(g_aiT_GPU+   16+k)*(*(src+   line))  	+ *(g_aiT_GPU+ 3*16+k)*(*(src+ 3*line)) +
			*(g_aiT_GPU+ 5*16+k)*(*(src+ 5*line)) 	+ *(g_aiT_GPU+ 7*16+k)*(*(src+ 7*line)) +
			*(g_aiT_GPU+ 9*16+k)*(*(src+ 9*line)) 	+ *(g_aiT_GPU+11*16+k)*(*(src+11*line)) +
			*(g_aiT_GPU+13*16+k)*(*(src+13*line)) 	+ *(g_aiT_GPU+15*16+k)*(*(src+15*line));
}

__device__ Void partialButterflyInverse16threadZeroB(\
		TCoeff* 	src, \
		Int 		line, \
		Int		k, \
		TCoeff* 	dst, \
		TMatrixCoeff *g_aiT_GPU)
{
	*(dst+k) = 	*(g_aiT_GPU+ 2*16+k)*(*(src+ 2*line)) + *(g_aiT_GPU+ 6*16+k)*(*(src+ 6*line)) +
			*(g_aiT_GPU+10*16+k)*(*(src+10*line)) + *(g_aiT_GPU+14*16+k)*(*(src+14*line));
}

__device__ Void partialButterflyInverse16threadZeroC(\
		TCoeff* 	src, \
		Int 		ai, \
		Int 		k, \
		Int 		bi, \
		Int 		line, \
		TCoeff* 	dst, \
		TMatrixCoeff *g_aiT_GPU)
{
	*(dst+k) = *(g_aiT_GPU+16*ai+k)*(*(src+ai*line )) + *(g_aiT_GPU+16*bi+k)*(*(src+bi*line ));
}

__device__ Void partialButterflyInverse16threadZero(\
		TCoeff*			src, \
		TCoeff*			dst, \
		Int 			shift, \
		Int 			line, \
		const TCoeff 	outputMinimum, \
		const TCoeff 	outputMaximum, \
		TCoeff 			add, \
		TMatrixCoeff *g_aiT_GPU)
{
	Int k;
	TCoeff E[8],O[8];
	TCoeff EE[4],EO[4];
	TCoeff EEE[2],EEO[2];

	/* Utilizing symmetry properties to the maximum to minimize the number of multiplications */
	for (k=0;k<8;k++)
	{
		partialButterflyInverse16threadZeroA(src, line, k, O, g_aiT_GPU);
	}

	for (k=0;k<4;k++)
	{
		partialButterflyInverse16threadZeroB(src, line, k, EO, g_aiT_GPU);
	}

	partialButterflyInverse16threadZeroC(src, 4, 0, 12, line, EEO, 	g_aiT_GPU);
	partialButterflyInverse16threadZeroC(src, 0, 0,  8, line, EEE, 	g_aiT_GPU);
	partialButterflyInverse16threadZeroC(src, 4, 1, 12, line, EEO, 	g_aiT_GPU);
	partialButterflyInverse16threadZeroC(src, 0, 1,  8, line, EEE, 	g_aiT_GPU);

	/* Combining even and odd terms at each hierarchy levels to calculate the final spatial domain vector */
	for (k=0;k<2;k++)
	{
		partialButterfly4ThreadOneP(EEE+k, 	EEO+k, 		EE+k);
		partialButterfly4ThreadOneN(EEE+1-k, 	EEO+1-k, 	EE+k+2);
	}

	for (k=0;k<4;k++)
	{
		partialButterfly4ThreadOneP(EE+k, 	EO+k, 	E+k);
		partialButterfly4ThreadOneN(EE+3-k, 	EO+3-k, E+k+4);
	}

	for (k=0;k<8;k++)
	{
		partialButterflyInverse4ThreadTwo(E[k], 	 O[k], 		dst+k, 		shift, outputMinimum, outputMaximum, add);
		partialButterflyInverse4ThreadTwo(E[7-k], 	-O[7-k], 	dst+k+8, 	shift, outputMinimum, outputMaximum, add);
	}
}

/** 16x16 inverse transform implemented using partial butterfly structure (1D)
 *  \param src            input data (transform coefficients)
 *  \param dst            output data (residual)
 *  \param shift          specifies right shift after 1D transform
 *  \param line
 *  \param outputMinimum  minimum for clipping
 *  \param outputMaximum  maximum for clipping
 */
__global__ Void partialButterflyInverse16(\
		TCoeff*			src, \
		TCoeff*			dst, \
		Int 			shift, \
		Int 			line, \
		const TCoeff 	outputMinimum, \
		const TCoeff 	outputMaximum, \
		TMatrixCoeff *g_aiT_GPU)
{
	__shared__ TCoeff add;
	Int j = blockIdx.x * blockDim.x + threadIdx.x;
	__syncthreads();

	if(j == 0) add = (shift > 0) ? (1<<(shift-1)) : 0;
	__syncthreads();

	if(j < 16)
	{
		partialButterflyInverse16threadZero(src+j, dst+16*j, shift, line, outputMinimum, outputMaximum, add, g_aiT_GPU);
	}
}



__device__ Void partialButterfly32threadZeroA(\
		TCoeff* src, \
		TCoeff* dst, \
		Int 	k, \
		Int 	line, \
		TCoeff 	add, \
		Int 	shift, \
		TMatrixCoeff *g_aiT_GPU)
{
	*(dst+k*line) = (*(g_aiT_GPU+32*k)*(*(src)) + *(g_aiT_GPU+32*k+1)*(*(src+1)) + add)>>shift;
}

__device__ Void partialButterfly32threadZeroB(\
		TCoeff* src, \
		TCoeff* dst, \
		Int 	k, \
		Int 	line, \
		TCoeff 	add, \
		Int 	shift, \
		TMatrixCoeff *g_aiT_GPU)
{
	*(dst+k*line) = (*(g_aiT_GPU+32*k)*(*(src)) + *(g_aiT_GPU+32*k+1)*(*(src+1)) +
			*(g_aiT_GPU+32*k+2)*(*(src+2)) + *(g_aiT_GPU+32*k+3)*(*(src+3)) + add)>>shift;
}

__device__ Void partialButterfly32threadZeroC(\
		TCoeff* src, \
		TCoeff* dst, \
		Int 	k, \
		Int 	line, \
		TCoeff 	add, \
		Int 	shift, \
		TMatrixCoeff *g_aiT_GPU)
{
	*(dst+k*line) = (*(g_aiT_GPU+32*k)*(*(src)) + *(g_aiT_GPU+32*k+1)*(*(src+1)) +
			*(g_aiT_GPU+32*k+2)*(*(src+2)) + *(g_aiT_GPU+32*k+3)*(*(src+3)) +
			*(g_aiT_GPU+32*k+4)*(*(src+4)) + *(g_aiT_GPU+32*k+5)*(*(src+5)) +
			*(g_aiT_GPU+32*k+6)*(*(src+6)) + *(g_aiT_GPU+32*k+7)*(*(src+7)) + add)>>shift;
}

__device__ Void partialButterfly32threadZeroD(\
		TCoeff* src, \
		TCoeff* dst, \
		Int 	k, \
		Int 	line, \
		TCoeff 	add, \
		Int 	shift, \
		TMatrixCoeff *g_aiT_GPU)
{
	*(dst+k*line) = (*(g_aiT_GPU+32*k)*(*(src)) + *(g_aiT_GPU+32*k+1)*(*(src+1)) +
			*(g_aiT_GPU+32*k+2)*(*(src+2)) 	+ *(g_aiT_GPU+32*k+3)*(*(src+3)) +
			*(g_aiT_GPU+32*k+4)*(*(src+4)) 	+ *(g_aiT_GPU+32*k+5)*(*(src+5)) +
			*(g_aiT_GPU+32*k+6)*(*(src+6)) 	+ *(g_aiT_GPU+32*k+7)*(*(src+7)) +
			*(g_aiT_GPU+32*k+8)*(*(src+8)) 	+ *(g_aiT_GPU+32*k+9)*(*(src+9)) +
			*(g_aiT_GPU+32*k+10)*(*(src+10)) 	+ *(g_aiT_GPU+32*k+11)*(*(src+11)) +
			*(g_aiT_GPU+32*k+12)*(*(src+12)) 	+ *(g_aiT_GPU+32*k+13)*(*(src+13)) +
			*(g_aiT_GPU+32*k+14)*(*(src+14)) 	+ *(g_aiT_GPU+32*k+15)*(*(src+15)) + add)>>shift;
}

__device__ Void partialButterfly32threadZero(\
		TCoeff* src, \
		TCoeff* dst, \
		Int 	shift, \
		Int 	line, \
		TCoeff 	add, \
		TMatrixCoeff *g_aiT_GPU)
{
	Int k;
	TCoeff E[16],O[16];
	TCoeff EE[8],EO[8];
	TCoeff EEE[4],EEO[4];
	TCoeff EEEE[2],EEEO[2];

	/* E and O*/
	for (k=0;k<16;k++)
	{
		partialButterfly4ThreadOneP(src+k, src+31-k, E+k);
		partialButterfly4ThreadOneN(src+k, src+31-k, O+k);
	}

	/* EE and EO */
	for (k=0;k<8;k++)
	{
		partialButterfly4ThreadOneP(E+k, E+15-k, EE+k);
		partialButterfly4ThreadOneN(E+k, E+15-k, EO+k);
	}

	/* EEE and EEO */
	for (k=0;k<4;k++)
	{
		partialButterfly4ThreadOneP(EE+k, EE+7-k, EEE+k);
		partialButterfly4ThreadOneN(EE+k, EE+7-k, EEO+k);
	}

	/* EEEE and EEEO */
	partialButterfly4ThreadOneP(EEE, EEE+3, EEEE);
	partialButterfly4ThreadOneN(EEE, EEE+3, EEEO);
	partialButterfly4ThreadOneP(EEE+1, EEE+2, EEEE+1);
	partialButterfly4ThreadOneN(EEE+1, EEE+2, EEEO+1);

	partialButterfly32threadZeroA(EEEE, dst,  0, line, add, shift, g_aiT_GPU);
	partialButterfly32threadZeroA(EEEE, dst, 16, line, add, shift, g_aiT_GPU);
	partialButterfly32threadZeroA(EEEO, dst,  8, line, add, shift, g_aiT_GPU);
	partialButterfly32threadZeroA(EEEO, dst, 24, line, add, shift, g_aiT_GPU);

	for (k=4;k<32;k+=8)
	{
		partialButterfly32threadZeroB(EEO, dst,  k, line, add, shift, g_aiT_GPU);
	}

	for (k=2;k<32;k+=4)
	{
		partialButterfly32threadZeroC(EO, dst,  k, line, add, shift, g_aiT_GPU);
	}

	for (k=1;k<32;k+=2)
	{
		partialButterfly32threadZeroD(O, dst,  k, line, add, shift, g_aiT_GPU);
	}
}

/** 32x32 forward transform implemented using partial butterfly structure (1D)
 *  \param src   input data (residual)
 *  \param dst   output data (transform coefficients)
 *  \param shift specifies right shift after 1D transform
 *  \param line
 */
__global__ Void partialButterfly32(TCoeff *src, TCoeff *dst, Int shift, Int line, \
		TMatrixCoeff *g_aiT_GPU)
{
	__shared__ TCoeff add;
	Int j = blockIdx.x * blockDim.x + threadIdx.x;
	__syncthreads();

	if(j == 0) add = (shift > 0) ? (1<<(shift-1)) : 0;
	__syncthreads();

	if(j < 32)
	{
		partialButterfly32threadZero(src+32*j, dst+j, shift, line, add, g_aiT_GPU);
	}
}



__device__ Void partialButterflyInverse32threadZeroA(\
		TCoeff* 		src, \
		Int 			k, \
		Int 			line, \
		TCoeff* 		dst, \
		TMatrixCoeff *g_aiT_GPU)
{
	*(dst + k) = 	*(g_aiT_GPU+k+32   )*(*(src+ line    )) + *(g_aiT_GPU+k+3*32 )*(*(src+ 3*line  )) +
			*(g_aiT_GPU+k+5*32 )*(*(src+ 5*line  )) + *(g_aiT_GPU+k+7*32 )*(*(src+ 7*line  )) +
			*(g_aiT_GPU+k+9*32 )*(*(src+ 9*line  )) + *(g_aiT_GPU+k+11*32)*(*(src+ 11*line )) +
			*(g_aiT_GPU+k+13*32)*(*(src+ 13*line )) + *(g_aiT_GPU+k+15*32)*(*(src+ 15*line )) +
			*(g_aiT_GPU+k+17*32)*(*(src+ 17*line )) + *(g_aiT_GPU+k+19*32)*(*(src+ 19*line )) +
			*(g_aiT_GPU+k+21*32)*(*(src+ 21*line )) + *(g_aiT_GPU+k+23*32)*(*(src+ 23*line )) +
			*(g_aiT_GPU+k+25*32)*(*(src+ 25*line )) + *(g_aiT_GPU+k+27*32)*(*(src+ 27*line )) +
			*(g_aiT_GPU+k+29*32)*(*(src+ 29*line )) + *(g_aiT_GPU+k+31*32)*(*(src+ 31*line ));
}

__device__ Void partialButterflyInverse32threadZeroB(\
		TCoeff* 		src, \
		Int 			k, \
		Int 			line, \
		TCoeff* 		dst, \
		TMatrixCoeff *g_aiT_GPU)
{
	*(dst + k) = 	*(g_aiT_GPU+k+2*32 )*(*(src+ 2*line  )) + *(g_aiT_GPU+k+6*32 )*(*(src+ 6*line  )) +
			*(g_aiT_GPU+k+10*32)*(*(src+ 10*line )) + *(g_aiT_GPU+k+14*32)*(*(src+ 14*line )) +
			*(g_aiT_GPU+k+18*32)*(*(src+ 18*line )) + *(g_aiT_GPU+k+22*32)*(*(src+ 22*line )) +
			*(g_aiT_GPU+k+26*32)*(*(src+ 26*line )) + *(g_aiT_GPU+k+30*32)*(*(src+ 30*line ));
}

__device__ Void partialButterflyInverse32threadZeroC(\
		TCoeff* 		src, \
		Int 			k, \
		Int 			line, \
		TCoeff* 		dst, \
		TMatrixCoeff *g_aiT_GPU)
{
	*(dst + k) = 	*(g_aiT_GPU+k+ 4*32)*(*(src+  4*line )) + *(g_aiT_GPU+k+12*32)*(*(src+ 12*line )) +
			*(g_aiT_GPU+k+20*32)*(*(src+ 20*line )) + *(g_aiT_GPU+k+28*32)*(*(src+ 28*line ));
}

__device__ Void partialButterflyInverse32threadZeroD(\
		TCoeff* 	src, \
		Int 		ai, \
		Int 		k, \
		Int 		bi, \
		Int 		line, \
		TCoeff* 	dst, \
		TMatrixCoeff *g_aiT_GPU)
{
	*(dst+k) = *(g_aiT_GPU+k+ai*32)*(*(src+ ai*line )) + *(g_aiT_GPU+k+bi*32)*(*(src+ bi*line ));
}


__device__ Void partialButterflyInverse32threadZero(\
		TCoeff* 		src, \
		TCoeff* 		dst, \
		Int 			shift, \
		Int 			line, \
		const TCoeff 	outputMinimum, \
		const TCoeff 	outputMaximum, \
		TCoeff 			add, \
		TMatrixCoeff *g_aiT_GPU)
{
	Int k;
	TCoeff E[16],O[16];
	TCoeff EE[8],EO[8];
	TCoeff EEE[4],EEO[4];
	TCoeff EEEE[2],EEEO[2];

	for (k=0;k<16;k++)
	{
		partialButterflyInverse32threadZeroA(src, k, line, O, g_aiT_GPU);
	}

	for (k=0;k<8;k++)
	{
		partialButterflyInverse32threadZeroB(src, k, line, EO, g_aiT_GPU);
	}

	for (k=0;k<4;k++)
	{
		partialButterflyInverse32threadZeroC(src, k, line, EEO, g_aiT_GPU);
	}

	partialButterflyInverse32threadZeroD(src, 8, 0, 24, line, EEEO, 	g_aiT_GPU);
	partialButterflyInverse32threadZeroD(src, 0, 0, 16, line, EEEE, 	g_aiT_GPU);
	partialButterflyInverse32threadZeroD(src, 8, 1, 24, line, EEEO, 	g_aiT_GPU);
	partialButterflyInverse32threadZeroD(src, 0, 1, 16, line, EEEE, 	g_aiT_GPU);

	/* Combining even and odd terms at each hierarchy levels to calculate the final spatial domain vector */
	partialButterfly4ThreadOneP(EEEE, 	EEEO,   EEE);
	partialButterfly4ThreadOneN(EEEE, 	EEEO,   EEE+3);
	partialButterfly4ThreadOneP(EEEE+1, 	EEEO+1, EEE+1);
	partialButterfly4ThreadOneN(EEEE+1, 	EEEO+1, EEE+2);

	for (k=0;k<4;k++)
	{
		partialButterfly4ThreadOneP(EEE+k, 	EEO+k,		EE+k);
		partialButterfly4ThreadOneN(EEE+3-k, 	EEO+3-k, 	EE+k+4);
	}

	for (k=0;k<8;k++)
	{
		partialButterfly4ThreadOneP(EE+k, 	EO+k,	E+k);
		partialButterfly4ThreadOneN(EE+7-k, 	EO+7-k, E+k+8);
	}

	for (k=0;k<16;k++)
	{
		partialButterflyInverse4ThreadTwo(E[k], 	 O[k], 		dst+k, 		shift, outputMinimum, outputMaximum, add);
		partialButterflyInverse4ThreadTwo(E[15-k], 	-O[15-k], 	dst+k+16, 	shift, outputMinimum, outputMaximum, add);
	}
}


/** 32x32 inverse transform implemented using partial butterfly structure (1D)
 *  \param src   input data (transform coefficients)
 *  \param dst   output data (residual)
 *  \param shift specifies right shift after 1D transform
 *  \param line
 *  \param outputMinimum  minimum for clipping
 *  \param outputMaximum  maximum for clipping
 */
__global__ Void partialButterflyInverse32(\
		TCoeff* 		src, \
		TCoeff* 		dst, \
		Int 			shift, \
		Int 			line, \
		const TCoeff 	outputMinimum, \
		const TCoeff 	outputMaximum, \
		TMatrixCoeff *g_aiT_GPU)
{
	__shared__ TCoeff add;
	Int j = blockIdx.x * blockDim.x + threadIdx.x;
	__syncthreads();

	if(j == 0) add = (shift > 0) ? (1<<(shift-1)) : 0;
	__syncthreads();

	if(j < 32)
	{
		partialButterflyInverse32threadZero(src+j, dst+32*j, shift, line, outputMinimum, outputMaximum, add, g_aiT_GPU);
	}
}

/** MxN forward transform (2D)
 *  \param bitDepth              [in]  bit depth
 *  \param block                 [in]  residual block
 *  \param coeff                 [out] transform coefficients
 *  \param iWidth                [in]  width of transform
 *  \param iHeight               [in]  height of transform
 *  \param useDST                [in]
 *  \param maxLog2TrDynamicRange [in]

 */
Void xTrMxN(Int bitDepth, TCoeff *block, TCoeff *coeff, Int iWidth, Int iHeight, Bool useDST, const Int maxLog2TrDynamicRange)
{
	TCoeff *block_GPU, *coeff_GPU, *tmp_GPU, *p;
	TMatrixCoeff *g_aiT_GPU, *pg;

	Int block_size = 0;

	const Int TRANSFORM_MATRIX_SHIFT = g_transformMatrixShift[TRANSFORM_FORWARD];
	const SChar		g_aucConvertToBit[ MAX_CU_SIZE+1 ] =
	{\
			-1, -1, -1, -1,  0, -1, -1, -1, \
			1, -1, -1, -1, -1, -1, -1, -1, \
			2, -1, -1, -1, -1, -1, -1, -1, \
			-1, -1, -1, -1, -1, -1, -1, -1, \
			3, -1, -1, -1, -1, -1, -1, -1, \
			-1, -1, -1, -1, -1, -1, -1, -1, \
			-1, -1, -1, -1, -1, -1, -1, -1, \
			-1, -1, -1, -1, -1, -1, -1, -1, \
			4};   // from width to log2(width)-2

	const Int shift_1st = ((g_aucConvertToBit[iWidth] + 2) +  bitDepth + TRANSFORM_MATRIX_SHIFT) - maxLog2TrDynamicRange;
	const Int shift_2nd = (g_aucConvertToBit[iHeight] + 2) + TRANSFORM_MATRIX_SHIFT;

	assert(shift_1st >= 0);
	assert(shift_2nd >= 0);

	switch (iWidth)
	{
	case 4:
		block_size = 16;
		hipMalloc(&block_GPU, 	16*sizeof(TCoeff));
		hipMalloc(&tmp_GPU, 	16*sizeof(TCoeff));
		hipMalloc(&g_aiT_GPU, 	16*sizeof(TMatrixCoeff));

		p = block;
		hipMemcpy(block_GPU, p, 16*sizeof(TCoeff), hipMemcpyHostToDevice);
		pg = (TMatrixCoeff*)&g_aiT4[TRANSFORM_FORWARD][0][0];
		hipMemcpy(g_aiT_GPU, pg, 16*sizeof(TMatrixCoeff), hipMemcpyHostToDevice);

		partialButterfly4<<<1,16>>>(block_GPU, tmp_GPU, shift_1st, iHeight, g_aiT_GPU);
		break;
	case 8:
		block_size = 64;
		hipMalloc(&block_GPU, 	64*sizeof(TCoeff));
		hipMalloc(&tmp_GPU, 	64*sizeof(TCoeff));
		hipMalloc(&g_aiT_GPU, 	64*sizeof(TMatrixCoeff));

		p = block;
		hipMemcpy(block_GPU, p, 64*sizeof(TCoeff), hipMemcpyHostToDevice);
		pg = (TMatrixCoeff*)&g_aiT8[TRANSFORM_FORWARD][0][0];
		hipMemcpy(g_aiT_GPU, pg, 64*sizeof(TMatrixCoeff), hipMemcpyHostToDevice);

		partialButterfly8<<<1,64>>>(block_GPU, tmp_GPU, shift_1st, iHeight, g_aiT_GPU);
		break;
	case 16:
		block_size = 256;
		hipMalloc(&block_GPU, 	256*sizeof(TCoeff));
		hipMalloc(&tmp_GPU, 	256*sizeof(TCoeff));
		hipMalloc(&g_aiT_GPU, 	256*sizeof(TMatrixCoeff));

		p = block;
		hipMemcpy(block_GPU, p, 256*sizeof(TCoeff), hipMemcpyHostToDevice);
		pg = (TMatrixCoeff*)&g_aiT16[TRANSFORM_FORWARD][0][0];
		hipMemcpy(g_aiT_GPU, pg, 256*sizeof(TMatrixCoeff), hipMemcpyHostToDevice);

		partialButterfly16<<<1,256>>>(block_GPU, tmp_GPU, shift_1st, iHeight, g_aiT_GPU);
		break;
	case 32:
		block_size = 1024;
		hipMalloc(&block_GPU, 	1024*sizeof(TCoeff));
		hipMalloc(&tmp_GPU, 	1024*sizeof(TCoeff));
		hipMalloc(&g_aiT_GPU, 	1024*sizeof(TMatrixCoeff));

		p = block;
		hipMemcpy(block_GPU, p, 1024*sizeof(TCoeff), hipMemcpyHostToDevice);
		pg = (TMatrixCoeff*)&g_aiT32[TRANSFORM_FORWARD][0][0];
		hipMemcpy(g_aiT_GPU, pg, 1024*sizeof(TMatrixCoeff), hipMemcpyHostToDevice);

		partialButterfly32<<<4,256>>>(block_GPU, tmp_GPU, shift_1st, iHeight, g_aiT_GPU);
		break;
	default:
		assert(0); exit (1); break;
	}
	hipDeviceSynchronize();

	hipMalloc(&coeff_GPU, 	block_size*sizeof(TCoeff));

	p = coeff;
	hipMemcpy(coeff_GPU, p, block_size*sizeof(TCoeff), hipMemcpyHostToDevice);

	switch (iHeight)
	{
	case 4:
		partialButterfly4<<<1,16>>>(tmp_GPU, coeff_GPU, shift_1st, iHeight, g_aiT_GPU);
		break;
	case 8:
		partialButterfly8<<<1,64>>>(tmp_GPU, coeff_GPU, shift_1st, iHeight, g_aiT_GPU);
		break;
	case 16:
		partialButterfly16<<<1,256>>>(tmp_GPU, coeff_GPU, shift_1st, iHeight, g_aiT_GPU);
		break;
	case 32:
		partialButterfly32<<<4,256>>>(tmp_GPU, coeff_GPU, shift_1st, iHeight, g_aiT_GPU);
		break;
	default:
		assert(0); exit (1); break;
	}
	hipDeviceSynchronize();

	hipMemcpy(coeff, coeff_GPU, block_size*sizeof(TCoeff), hipMemcpyDeviceToHost);

	hipFree(block_GPU);
	hipFree(coeff_GPU);
	hipFree(tmp_GPU);
	hipFree(g_aiT_GPU);
}


/** MxN inverse transform (2D)
 *  \param bitDepth              [in]  bit depth
 *  \param coeff                 [in]  transform coefficients
 *  \param block                 [out] residual block
 *  \param iWidth                [in]  width of transform
 *  \param iHeight               [in]  height of transform
 *  \param useDST                [in]
 *  \param maxLog2TrDynamicRange [in]
 */
Void xITrMxN(Int bitDepth, TCoeff *coeff, TCoeff *block, Int iWidth, Int iHeight, Bool useDST, const Int maxLog2TrDynamicRange)
{
	TCoeff *block_GPU, *coeff_GPU, *tmp_GPU, *p;
	TMatrixCoeff *g_aiT_GPU, *pg;

	Int block_size = 0;

	const Int TRANSFORM_MATRIX_SHIFT = g_transformMatrixShift[TRANSFORM_INVERSE];

	Int shift_1st = TRANSFORM_MATRIX_SHIFT + 1; //1 has been added to shift_1st at the expense of shift_2nd
	Int shift_2nd = (TRANSFORM_MATRIX_SHIFT + maxLog2TrDynamicRange - 1) - bitDepth;
	const TCoeff clipMinimum = -(1 << maxLog2TrDynamicRange);
	const TCoeff clipMaximum =  (1 << maxLog2TrDynamicRange) - 1;

	assert(shift_1st >= 0);
	assert(shift_2nd >= 0);

	switch (iHeight)
	{
	case  4:
		block_size = 16;
		hipMalloc(&coeff_GPU, 	16*sizeof(TCoeff));
		hipMalloc(&tmp_GPU, 	16*sizeof(TCoeff));
		hipMalloc(&g_aiT_GPU, 	16*sizeof(TMatrixCoeff));

		p = coeff;
		hipMemcpy(coeff_GPU, p, 16*sizeof(TCoeff), hipMemcpyHostToDevice);
		pg = (TMatrixCoeff*)&g_aiT4[TRANSFORM_INVERSE][0][0];
		hipMemcpy(g_aiT_GPU, pg, 16*sizeof(TMatrixCoeff), hipMemcpyHostToDevice);

		partialButterflyInverse4<<<1,16>>>(coeff_GPU, tmp_GPU, shift_1st, iWidth, clipMinimum, clipMaximum, g_aiT_GPU);
		break;
	case  8:
		block_size = 64;
		hipMalloc(&coeff_GPU, 	64*sizeof(TCoeff));
		hipMalloc(&tmp_GPU, 	64*sizeof(TCoeff));
		hipMalloc(&g_aiT_GPU, 	64*sizeof(TMatrixCoeff));

		p = coeff;
		hipMemcpy(coeff_GPU, p, 64*sizeof(TCoeff), hipMemcpyHostToDevice);
		pg = (TMatrixCoeff*)&g_aiT8[TRANSFORM_INVERSE][0][0];
		hipMemcpy(g_aiT_GPU, pg, 64*sizeof(TMatrixCoeff), hipMemcpyHostToDevice);

		partialButterflyInverse8<<<1,64>>>(coeff_GPU, tmp_GPU, shift_1st, iWidth, clipMinimum, clipMaximum, g_aiT_GPU);
		break;
	case 16:
		block_size = 256;
		hipMalloc(&coeff_GPU, 	256*sizeof(TCoeff));
		hipMalloc(&tmp_GPU, 	256*sizeof(TCoeff));
		hipMalloc(&g_aiT_GPU, 	256*sizeof(TMatrixCoeff));

		p = coeff;
		hipMemcpy(coeff_GPU, p, 256*sizeof(TCoeff), hipMemcpyHostToDevice);
		pg = (TMatrixCoeff*)&g_aiT16[TRANSFORM_INVERSE][0][0];
		hipMemcpy(g_aiT_GPU, pg, 256*sizeof(TMatrixCoeff), hipMemcpyHostToDevice);

		partialButterflyInverse16<<<1,256>>>(coeff_GPU, tmp_GPU, shift_1st, iWidth, clipMinimum, clipMaximum, g_aiT_GPU);
		break;
	case 32:
		block_size = 1024;
		hipMalloc(&coeff_GPU, 	1024*sizeof(TCoeff));
		hipMalloc(&tmp_GPU, 	1024*sizeof(TCoeff));
		hipMalloc(&g_aiT_GPU, 	1024*sizeof(TMatrixCoeff));

		p = coeff;
		hipMemcpy(coeff_GPU, p, 1024*sizeof(TCoeff), hipMemcpyHostToDevice);
		pg = (TMatrixCoeff*)&g_aiT32[TRANSFORM_INVERSE][0][0];
		hipMemcpy(g_aiT_GPU, pg, 1024*sizeof(TMatrixCoeff), hipMemcpyHostToDevice);

		partialButterflyInverse32<<<4,256>>>(coeff_GPU, tmp_GPU, shift_1st, iWidth, clipMinimum, clipMaximum, g_aiT_GPU);
		break;
	default:
		assert(0); exit (1); break;
	}
	hipDeviceSynchronize();

	hipMalloc(&block_GPU, 	block_size*sizeof(TCoeff));

	p = block;
	hipMemcpy(block_GPU, p, block_size*sizeof(TCoeff), hipMemcpyHostToDevice);

	switch (iWidth)
	{
	// Clipping here is not in the standard, but is used to protect the "Pel" data type into which the inverse-transformed samples will be copied
	case  4:
		partialButterflyInverse4<<<1,16>>>(tmp_GPU, block_GPU, shift_2nd, iHeight, std::numeric_limits<Pel>::min(), std::numeric_limits<Pel>::max(), g_aiT_GPU);
		break;
	case  8:
		partialButterflyInverse8<<<1,64>>>(tmp_GPU, block_GPU, shift_2nd, iHeight, std::numeric_limits<Pel>::min(), std::numeric_limits<Pel>::max(), g_aiT_GPU);
		break;
	case 16:
		partialButterflyInverse16<<<1,256>>>(tmp_GPU, block_GPU, shift_2nd, iHeight, std::numeric_limits<Pel>::min(), std::numeric_limits<Pel>::max(), g_aiT_GPU);
		break;
	case 32:
		partialButterflyInverse32<<<4,256>>>(tmp_GPU, block_GPU, shift_2nd, iHeight, std::numeric_limits<Pel>::min(), std::numeric_limits<Pel>::max(), g_aiT_GPU);
		break;
	default:
		assert(0); exit (1); break;
	}
	hipDeviceSynchronize();

	hipMemcpy(block, block_GPU, block_size*sizeof(TCoeff), hipMemcpyDeviceToHost);

	hipFree(block_GPU);
	hipFree(coeff_GPU);
	hipFree(tmp_GPU);
	hipFree(g_aiT_GPU);
}

DCTHEVCdct::DCTHEVCdct() {
	initROM();
}
